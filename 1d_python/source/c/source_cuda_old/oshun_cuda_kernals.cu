#include "hip/hip_runtime.h"
// kernals
/*
	Version 4:
		Cleanup up code... made it so that there were no register spillings or stack frmae bytes genrated as reported by PTAX.. This is the baseline version
		for the furthur cleanups.

	Version 5:
		Try to minimize all mem access... calculated most everything in the from scratch in the kernal.. Only question is how well looks work for the
		p bin values... should they be repeated for each thread? should there be 1 copy? will it be cached?
*/
#include <iostream>
#include <vector>
#include <valarray>
#include <complex>
//#include <stdint.h>

//#include <mpi.h>
#include <math.h>
#include <stdio.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <thrust/scan.h>
#include "matrices.h"

// Interface 
//#include "decl-input.h"
//#include "decl-state.h"
//#include "decl-fokkerplanck.h"

//#include "oshun_cuda.h"
#include "oshun_cuda.cuh"
#define FP_TYPE double
#define DATA double *
#define uint unsigned int
#define M_PI 3.14159265358979323846


extern "C" void set_FP_constants(int _leng, FP_TYPE _c_kpre, uint _NB, FP_TYPE _density_np);
extern "C" void simple_test_kernal();
extern "C" void add_1_to_all();
extern "C" void cudasafe( hipError_t error, const char *message);
extern "C" void scan_test();
extern "C" void scan_test_4_at_a_time();
extern "C" void eval_test(	double * fc, double * dest, double *vr, 
							double * U1, double *U1m1, double * U2, double *U2m1,
							double * U3,
							double * U4, double *U4m1,
							double * Pn, double *Qn,
							uint size, int debug);

extern "C" void test_scans(double * crap);

extern "C" void eval_rk4_v6 (	double * fc, double * dest, double *vr, 
							//double * U1, double *U1m1, double * U2, double *U2m1,
							//double * U3,
							//double * U4, double *U4m1,
							//double * Pn, double *Qn,
							double * __restrict__ cell_data__precomp1,   // these should be the size of the number of cells.*pr
							double * __restrict__ cell_data__precomp2,
							double * __restrict__ cell_data__precomp3,
							double * __restrict__ cell_data__precomp4,
							double * __restrict__ cell_data__precomp6,
							uint size, int padded_size, int debug, double h, int numh, int num_cells_x, int num_cells_y,
							const double * __restrict__ U4,
									const double * __restrict__ U4m1,
									const double * __restrict__ U2,
									const double * __restrict__ U2m1);
								
//extern GpuProperties gpu_properties;

#define id threadIdx.x
#define bi blockIdx.x == 0 &&

#define cutilCheckMsg(msg)           __cutilGetLastError (msg, __FILE__, __LINE__)

inline void __cutilGetLastError( const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        printf("%s(%i) : cutilCheckMsg() CUTIL CUDA error : %s : (%d) %s.\n", file, line, errorMessage, (int)err, 
			hipGetErrorString( err ) );
        exit(-1);
    }
} 

// constants for FP evaluatoin...
__constant__ int leng;
__constant__ FP_TYPE c_kpre;
__constant__ uint NB;
__constant__ double density_np;
__device__ double LOG_LAMBDA;

void set_FP_constants(int _leng, FP_TYPE _c_kpre, uint _NB, FP_TYPE _density_np) {

	hipMemcpyToSymbol(HIP_SYMBOL("leng"), &_leng, 4, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("c_kpre"), &_c_kpre, 8, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("NB"), &_NB, 4, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("density_np"), &_density_np, 8, 0, hipMemcpyHostToDevice);
}



inline __device__ double LOGee_cuda__v2(double ne, double Te) {
		
        double lnee;
        
//      if the density is positive
        if (ne > 0.000000001) {
            Te /= (3.0*ne);
            Te *= 511000; // Temperature in eV
            ne *= density_np;

            Te = (double) __logf((float) Te); 
            ne = (double) __logf((float) ne);
            lnee = 23.5 - 0.5*ne + 1.25*Te - sqrt(0.00001+0.0625*(Te-2.0)*(Te-2.0));

            if (lnee > 2.0) return lnee;
        }
        // Default minimum "2"

        return 2.0; 
}

inline __device__ double LOGee_cuda(double ne, double Te) {
//-------------------------------------------------------------------
//   Calculate the Coulomb logarithm for electron-electron collisions
//-------------------------------------------------------------------
//      Note: that the results here assume the distribution functions
//      are nonrelativistic, as is the case for the rest of the F-P 
//      part of the code.

        double lnee;
        
//      if the density is positive
        if (ne > 0.000000001) {
            Te /= (3.0*ne);
            Te *= 511000; // Temperature in eV
            ne *= density_np;

            Te = log(Te); 
            ne = log(ne);
            lnee = 23.5 - 0.5*ne + 1.25*Te - sqrt(0.00001+0.0625*(Te-2.0)*(Te-2.0));

            if (lnee > 2.0) return lnee;
        }
        // Default minimum "2"

        return 2.0; 
}


// TODO: this can be loads better
__device__  double G_cuda(const int n, const DATA fin,
					const DATA vr, const FP_TYPE J1) {
	double i2s, i4s;
	double f00( (fin[0] - fin[1]*(vr[0]*vr[0])/(vr[1]*vr[1]))/ (1.0 - (vr[0]*vr[0])/(vr[1]*vr[1])) );
	//printf("\tf00 %e\n", f00);
	i2s = f00*pow(vr[n],3)/3.0 + (fin[1]-f00)*pow(vr[n],5)/(vr[1]*vr[1])*0.2;
	//printf("\ti2s %e\n", i2s);
	i4s = f00*pow(vr[n],5)*0.2 + (fin[1]-f00)*pow(vr[n],7)/(vr[1]*vr[1]*7.0);
	//printf("\ti4s %e\n", i4s);
	return fin[n]*i4s + (pow(vr[n],3)*fin[n]-3.0*i2s) * J1;
}


////////////////////////////////////////////////////////////////////////////////
// Scan kernels double
////////////////////////////////////////////////////////////////////////////////
// orginall 256
#define THREADBLOCK_SIZE 512
#define LOG2_WARP_SIZE 5U
#define WARP_SIZE (1U << LOG2_WARP_SIZE)

// REVERSE SCAN KERNALS
//-------------------------------------------------------------------------------------------------
	//Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
	//assuming size <= WARP_SIZE
	inline __device__ double warpScanInclusive_rev(double idata, volatile double *s_Data, uint size) {
		// this genrates a a sequence of [0,size][2*(size+1),(3*size-1)]
		
		//uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
		
		// diff
		uint pos = (2*threadIdx.x - 3*(threadIdx.x & (size - 1))) + size - 1;
		s_Data[pos] = 0.0;
		
		//pos += (2*(size-threadIdx.x) -1); // DUMB TODO: ADAM FACTOR THIS BETTER BUT I AM TIRED //size;
		//pos = 2*size - 1 - pos;
		pos += size;
		s_Data[pos] = idata;
		
		//printf("Thread %d (size %d) mapped to %d (%d)\n", threadIdx.x, size, pos, pos-size);
		
		for(uint offset = 1; offset < size; offset <<= 1) {
			s_Data[pos] += s_Data[pos - offset];
		}
		
		return s_Data[pos];
	}

	inline __device__ double warpScanExclusive_rev(double idata, volatile double *s_Data, uint size) {
		return warpScanInclusive_rev(idata, s_Data, size) - idata;
	}

	inline __device__ double scan1Inclusive_rev(double idata, volatile double *s_Data, uint size) {
		
		if(size > WARP_SIZE) {
			//Bottom-level inclusive warp scan
			double warpResult = warpScanInclusive_rev(idata, s_Data, WARP_SIZE);
			
			//Save top elements of each warp for exclusive warp scan
			//sync to wait for warp scans to complete (because s_Data is being overwritten)
			__syncthreads();
			
			//if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) ) {
			if( ((threadIdx.x-1) & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )  // so ghetto
				s_Data[threadIdx.x >> LOG2_WARP_SIZE] = warpResult;
			
			// wait for warp scans to complete
			__syncthreads();
			if( threadIdx.x < (THREADBLOCK_SIZE / WARP_SIZE) ) {
				//grab top warp elements
				double val = s_Data[threadIdx.x];
				//calculate exclsive scan and write back to shared memory
				s_Data[threadIdx.x] = warpScanExclusive_rev(val, s_Data, size >> LOG2_WARP_SIZE);
			}

			//return updated warp scans with exclusive scan results
			__syncthreads();
			return warpResult + s_Data[threadIdx.x >> LOG2_WARP_SIZE];
		} else { 
			return warpScanInclusive_rev(idata, s_Data, size);
		}
	}

	inline __device__ double scan1Exclusive_rev(double idata, volatile double *s_Data, uint size){
		return scan1Inclusive_rev(idata, s_Data, size) - idata;
	}
// Simple Forward Scan Kernal
//-------------------------------------------------------------------------------------------------
	// simple 128
	inline __device__ double warpScanInclusiveSimple(double idata, double *s_Data, uint size) {
		
		s_Data[threadIdx.x] = idata;
		
		if(threadIdx.x > 0) s_Data[threadIdx.x] += s_Data[threadIdx.x - 1];
		__syncthreads();
		if(threadIdx.x > 1) s_Data[threadIdx.x] += s_Data[threadIdx.x - 2];
		__syncthreads();
		if(threadIdx.x > 3) s_Data[threadIdx.x] += s_Data[threadIdx.x - 4];
		__syncthreads();
		if(threadIdx.x > 7) s_Data[threadIdx.x] += s_Data[threadIdx.x - 8];
		__syncthreads();
		if(threadIdx.x > 15) s_Data[threadIdx.x] += s_Data[threadIdx.x - 16];
		__syncthreads();
		if(threadIdx.x > 31) s_Data[threadIdx.x] += s_Data[threadIdx.x - 32];
		__syncthreads();
		if(threadIdx.x > 63) s_Data[threadIdx.x] += s_Data[threadIdx.x - 64];
		__syncthreads();
		
		//if( blockIdx.x == 0 ) {
		//	printf("|%d:%d: %e| \n", blockIdx.x, threadIdx.x, s_Data[threadIdx.x]);
		//}
		return s_Data[threadIdx.x];
	}
	
	// assume 128 for now... and WARP_SIZE=32
	inline __device__ double warpScanInclusiveSimpleWarp(double idata, volatile double *s_Data , uint size) {   //, volatile double *s_data__warp_results, uint size) {
		
		s_Data[threadIdx.x] = idata;
		__syncthreads(); // suspect
		
		//uint warp_index = (threadIdx.x & (size - 1));
		uint warp_index = (threadIdx.x & 31);
		
		if(warp_index > 0) s_Data[threadIdx.x] += s_Data[threadIdx.x - 1];
		//__syncthreads();
		if(warp_index > 1) s_Data[threadIdx.x] += s_Data[threadIdx.x - 2];
		//__syncthreads();
		if(warp_index > 3) s_Data[threadIdx.x] += s_Data[threadIdx.x - 4];
		//__syncthreads();
		if(warp_index > 7) s_Data[threadIdx.x] += s_Data[threadIdx.x - 8];
		//__syncthreads();
		if(warp_index > 15) s_Data[threadIdx.x] += s_Data[threadIdx.x - 16];
		__syncthreads();
		
		double intra_warp_result = s_Data[threadIdx.x];
		
		//return intra_warp_result;
		
		if(warp_index == 31) {
			s_Data[threadIdx.x >> LOG2_WARP_SIZE] = intra_warp_result;
		}
		__syncthreads();
		
		if(threadIdx.x < 5) {
			double my_val = s_Data[threadIdx.x];
			
			if(threadIdx.x > 0) s_Data[threadIdx.x] += s_Data[threadIdx.x - 1];
			if(threadIdx.x > 1) s_Data[threadIdx.x] += s_Data[threadIdx.x - 2];
			s_Data[threadIdx.x] -= my_val;
		}
		__syncthreads();
		
		
		return s_Data[threadIdx.x >> LOG2_WARP_SIZE] + intra_warp_result; 
	}

		// assume 128 for now... and WARP_SIZE=32
	inline __device__ double warpScanInclusiveSimpleWarp_rev(double idata, volatile double *s_Data , uint size) {   //, volatile double *s_data__warp_results, uint size) {
		
		__syncthreads();
		s_Data[threadIdx.x] = idata;
		__syncthreads(); // suspect
		
		
		
		//uint warp_index = (threadIdx.x & (size - 1));
		uint warp_index = (threadIdx.x & 31);
		__syncthreads();
		if(warp_index < 31) s_Data[threadIdx.x] += s_Data[threadIdx.x + 1];
		__syncthreads();
		if(warp_index < 30) s_Data[threadIdx.x] += s_Data[threadIdx.x + 2];
		__syncthreads();
		if(warp_index < 28) s_Data[threadIdx.x] += s_Data[threadIdx.x + 4];
		__syncthreads();
		if(warp_index < 24) s_Data[threadIdx.x] += s_Data[threadIdx.x + 8];
		__syncthreads();
		if(warp_index < 16) s_Data[threadIdx.x] += s_Data[threadIdx.x + 16];
		__syncthreads();
		
		double intra_warp_result = s_Data[threadIdx.x];
		__syncthreads();
		
		if(warp_index == 0) {
			s_Data[threadIdx.x >> LOG2_WARP_SIZE] = intra_warp_result;
		}
		__syncthreads();
		
		if(threadIdx.x < 4) {
			double my_val = s_Data[threadIdx.x];
			
			if(threadIdx.x < 3) s_Data[threadIdx.x] += s_Data[threadIdx.x + 1];
			if(threadIdx.x < 2) s_Data[threadIdx.x] += s_Data[threadIdx.x + 2];
			s_Data[threadIdx.x] -= my_val;
			
			//printf("%d %e \n", threadIdx.x, s_Data[threadIdx.x]);
		}
		__syncthreads();
		
		return s_Data[threadIdx.x >> LOG2_WARP_SIZE] + intra_warp_result; 
	}

// FORWARD SCAN KERNALS
//-------------------------------------------------------------------------------------------------
	// Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
	// assuming: size <= WARP_SIZE
	inline __device__ double warpScanInclusive(double idata, volatile double *s_Data, uint size){
		// this genrates a a sequence of [0,size][2*(size+1),(3*size-1)]
		
		/*
		 If 'size' is a power of 2 such that 'size' = 2^p then ('size'-1) is a bit pattern of all 1's for the p least significant bits. and all zeros for all
		    higher bits (for example if the '=size of an interger is 2^32 and 'size' is 2^8 then ('size'-1) = 00000000000000000000000011111111
		    so '(threadIdx.x & (size - 1)' will effectivly cause all bits in 'threadIdx.x ' above p to be zeroed and will pass all lower p bits of 'threadIdx.x ' 
		    to be passed though unaltered. This is another way to write '(threadIdx.x mod 'size')' which gives a sequence of [0....(size-1)] repeated.
		   (we have to so this because Mod is pathalogically slow on Cuda at this point... cuda is terrible at integer aritmetic.)
		
		 So '2 * threadIdx.x - (threadIdx.x & (size - 1))' does the following mapping (assuming size==WARP_SIZE but threadIdx.x can be > WARP_SIZE):
		              0  <= threadIdx.x <  1*WARP_SIZE  maps to  0          ..( 1*WARP_SIZE - 1 )     for WARP_SIZE 32:   0   .. 31
		      WARP_SIZE  <= threadIdx.x <  2*WARP_SIZE  maps to  2*WARP_SIZE..( 3*WARP_SIZE - 1 )     for WARP_SIZE 32:   64  .. 95
		    2*WARP_SIZE  <= threadIdx.x <  3*WARP_SIZE  maps to  4*WARP_SIZE..( 5*WARP_SIZE - 1 )     for WARP_SIZE 32:   128 .. 159
		    3*WARP_SIZE  <= threadIdx.x <  4*WARP_SIZE  maps to  6*WARP_SIZE..( 7*WARP_SIZE - 1 )     for WARP_SIZE 32:   192 .. 223
		    4*WARP_SIZE  <= threadIdx.x <  5*WARP_SIZE  maps to  8*WARP_SIZE..( 9*WARP_SIZE - 1 )     for WARP_SIZE 32:   256 .. 287
		    ....
		   15*WARP_SIZE  <= threadIdx.x < 16*WARP_SIZE  maps to 30*WARP_SIZE..(31*WARP_SIZE - 1 )     for WARP_SIZE 32:   960 .. 991
		*/
		uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
		s_Data[pos] = 0;
		pos += size;
		s_Data[pos] = idata;
		/*
			so the above statements create a structure that looks like (for warp of 32)
			
			
			buffer index: |---000 to 031---|---032 to 063---|---064 to 095---|---096 to 127---|---128 to 159---|---160 to 191---|
			      values: |        0       |vals 000 to 031 |        0       |vals 032 to 063 |        0       |vals 064 to 095 |
			
			where "vals 000 to 031" means the values passed in by the threads with 000 < threadIdx.x < 031 written in consecutive memory locations.
			
		*/
		
		// in this routine size <= WARP_SIZE...
		// the following statements are a bit magical... because they are GAURENTEED to executed simultaneously. It's kinda freeky.
		// Assume size == 32:  [X] = current value of index location X
		//                     |x| = value that was at index x at start of loop
		//                     {x} = value passed ib by thread with threadIdx.x = x 
		//                     |x..y| = sum all values inclusive of x any y that were there at start of loop
		//                     {x..y} = sum all values inclusive of values passed in thread with threadIdx.x={x to y}
		/*
			1st time though for loop (offset == 1):
				index[32] = |32| + |31| == {0} + 0
				index[33] = |33| + |32| == {1} + {0}
				index[34] = |34| + [33] == {2} + {1}
				index[35] = |35| + [34] == {3} + {2}
				index[36] = |36| + [35] == {4} + {3}
				index[62] = |62| + [61] == {30} + {29}
				index[63] = |63| + |62| == {31} + {30}
			2nd time though for loop (offset == 2):
				index[32] = [32] + [30] = (|32| + |31|) + (0) == {0} + 0 + 0 
				index[33] = [33] + [31] = (|33| + |32|) + (0) == {0} + {1} + 0 
				index[34] = [34] + [32] = (|34| + |33|) + (|32| + |31|) == {2} + {1} + {0} + 0
				index[35] = [35] + [33] = (|35| + [34]) + (|33| + |32|) == {3} + {2} + {1} + {1}
				index[36] = [36] + [34] = (|36| + [35]) + (|34| + [33]) == {4} + {3} + {2} + {1}
				index[62] = [62] + [60] = (|62| + [61]) + (|60| + [59]) == {30} + {29} + {28} + {27}
				index[63] = [63] + [61] = (|63| + |62|) + (|62| + [61]) == {31} + {30} + {29} + {28}
			3rd time through for loop (offset == 4):
				index[32] = [32] + [28] = [(|32| + |31|) + (0) ] + (0) == [{0} + 0 + 0 + 0] + [0 + 0 + 0 + 0]
				index[33] = [33] + [29] = [(|33| + |32|) + (0) ] + (0) == [{1} + {0} + 0 + 0] + [0 + 0 + 0 + 0]
				index[34] = [34] + [30] = [(|34| + |33|) + (|32| + |31|)] + [(0)] = [{2}+{1}+{0} + 0] + [0 + 0 + 0 + 0]
				index[35] = [35] + [31] = [(|35| + [34]) + (|33| + |32|)] + [(0)] = [{3}+{2}+{1}+{0}] + [0 + 0 + 0 + 0]
				index[36] = [36] + [32] = [(|36| + [35]) + (|34| + [33])] + [(|32| + |31|) + (0)]  = [{4}+{3}+{2}+{1}]+[{0} + 0 + 0 + 0]
				index[62] = [62] + [58] = [(|62| + [61]) + (|60| + [59])] + [(|58| + [57]) + (|56| + [55])] = [{30} + {29} + {28} + {27}] + [{26} + {25} + {24} + {23}]
				index[63] = [63] + [59] = [(|63| + |62|) + (|62| + [61])] + [(|59| + [58]) + (|57| + [56])] = [{31} + {30} + {29} + {28}] + [{27} + {26} + {25} + {24}]
			4th time through for loop (offset == 8):
				index[32] = [32] + [24] = [32] + 0 = {0}
				index[33] = [33] + [25] = [33] + 0 = {0..1}
				index[34] = [34] + [26] = [34] + 0 = {0..2}
				index[35] = [35] + [27] = [35] + 0 = {0..3}
				index[36] = [36] + [28] = [36] + 0 = {0..4}
				index[62] = [62] + [54] = [|62..55|] + [|54..47|] = {30..23} + {22..15} = {30..15}
				index[63] = [63] + [55] = [|63..56|] + [|55..48|] = {31..24} + {23..16} = {31..16}
			5th time through for loop (offset == 16):
				index[32] = {0}
				index[33] = {0..1}
				index[34] = {0..2}
				index[35] = {0..3}
				index[36] = {0..4}
				index[62] = [62] + [46] = [|62..47|] + [|46..30|] = {30..15} + {14..0} = {30..0}
				index[63] = [63] + [47] = [|63..48|] + [|47..31|] = {31..16} + {15..0} = {31..0}
			Done.
		*/
		for(uint offset = 1; offset < size; offset <<= 1)
			s_Data[pos] += s_Data[pos - offset];
		
		return s_Data[pos];
	}
	
	inline __device__ double warpScanExclusive(double idata, volatile double *s_Data, uint size){
		return warpScanInclusive(idata, s_Data, size) - idata;
	}
	
	inline __device__ double scan1Inclusive(double idata, volatile double *s_Data, uint size) {
		
		if(size > WARP_SIZE) {
			//Bottom-level inclusive warp scan
			double warpResult = warpScanInclusive(idata, s_Data, WARP_SIZE);
			
			// first wait for all treads to finish
			// since s_data is being written.. 
			// and we need it in it's final state before moving on.
			__syncthreads();
			
			// --------------------------------------------------------
			//Save top elements of each warp for exclusive warp scan
			//
			// this if statement is only true for the 'top' threadIdx.x in each warp
			//    so WARP_SIZE=32, the if is true for threads with threadIdx.x = [31,63, 95, 127, 159, 191, 223, 255, 287, 319, 351, 383, 415, 447, 479, 511]
			// and writes the value to index in s_data of                      = [0 , 1,  2,   3,   4,   5,   6,   7,   8,   9,  10,  11,  12,  13,  14,  15]
			if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
				s_Data[threadIdx.x >> LOG2_WARP_SIZE] = warpResult;
			
			//wait for warp scans to complete
			__syncthreads();
			
			// Done saving the 'top' warp results to the s_data buffer
			// --------------------------------------------------------
			
			
			//--------------------------------------------------------
			// Add the 'top' results to get final result.
			//
			// for all threads with threadinx.x = 0 to 15  [one thread for every 'top' result written in previous result].
			// Do another 'warp scan' but with values pass in that are the sum of a Warp.
			if( threadIdx.x < (THREADBLOCK_SIZE / WARP_SIZE) ) {
				//grab top warp elements
				double val = s_Data[threadIdx.x];
				//printf("FUCK %d: %e \n", threadIdx.x,  val);
				//calculate exclsive scan and write back to shared memory
				s_Data[threadIdx.x] = warpScanExclusive(val, s_Data, size >> LOG2_WARP_SIZE);
			}
			
			/* so now the first 16 elements (THREADBLOCK_SIZE / WARP_SIZE elements in general... 16 for 512/32) of s_data have:
				s_data[0] =  warps(0..0) =  {0..31}
				s_data[1] =  warps(0..1) =  {0..63}
				s_data[2] =  warps(0..2) =  {0..95}
				s_data[3] =  warps(0..3) =  {0..127}
				s_data[4] =  warps(0..4) =  {0..159}
				s_data[14] = warps(0..14) = {0..479}
				s_data[15] = warps(0..15) = {0..511}
			*/
			// wait for all write to s_data to be done bewfore moving on.
			__syncthreads();
			
			// Done adding top results...
			//--------------------------------------------------------
			
			
			// Now all threads partisipate again...
			// The final result passed back is the "preceeding warp sum" plus the sum within warp that thread is in.!
			
			
			return warpResult + s_Data[threadIdx.x >> LOG2_WARP_SIZE];
			
		} else {
			
			return warpScanInclusive(idata, s_Data, size);
			
		}
	}
	
	inline __device__ double scan1Exclusive(double idata, volatile double *s_Data, uint size) {
		return scan1Inclusive(idata, s_Data, size) - idata;
	}
//-------------------------------------------------------------------------------------------------


inline __device__ double4 scan4Inclusive(double4 idata4, volatile double *s_Data, uint size) {
	
	//Level-0 inclusive scan
	idata4.y += idata4.x;
	idata4.z += idata4.y;
	idata4.w += idata4.z;
	
	//Level-1 exclusive scan
	double oval = scan1Exclusive(idata4.w, s_Data, size / 4);
	
	idata4.x += oval;
	idata4.y += oval;
	idata4.z += oval;
	idata4.w += oval;
	
	return idata4;
}

inline __device__ double4 scan4Exclusive(double4 idata4, volatile double *s_Data, uint size){
	double4 odata4 = scan4Inclusive(idata4, s_Data, size);
	odata4.x -= idata4.x;
	odata4.y -= idata4.y;
	odata4.z -= idata4.z;
	odata4.w -= idata4.w;
	return odata4;
}

__global__ void scanInclusiveShared(
    double4 *d_Dst,
    double4 *d_Src,
    uint size
){
    __shared__ double s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    double4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    //uint4 odata4 = scan4Exclusive(idata4, s_Data, size);
	double4 odata = scan4Inclusive(idata4, s_Data, size);

    //Write back
    d_Dst[pos] = odata;
}

__global__ void scanInclusiveShared_one_at_a_time(	double *d_Dst,
													double *d_Src,
													uint size){
	__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	uint pos = blockIdx.x * blockDim.x + threadIdx.x;
	double data = d_Src[pos];
	double final_val = scan1Inclusive(data, s_Data, size);

	//data = 1.4;
	double final_val2 = scan1Inclusive_rev(data, s_Data, size);
	d_Dst[pos] = final_val2;

}

__global__ void scanInclusiveShared_one_at_a_time_rev(	double *d_Dst,
													double *d_Src,
													uint size){
	__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	uint pos = blockIdx.x * blockDim.x + threadIdx.x;
	
	double data = d_Src[pos];
	double final_val = scan1Inclusive_rev(data, s_Data, size);
	d_Dst[pos] = final_val;

}


void scan_test() {
	int size = 512;
	int size_logical = 512;

	// cheat for now and bump up mem size so eveything is even.
	int blocks_needed = size / (THREADBLOCK_SIZE);
	if((size % (THREADBLOCK_SIZE)) != 0) {
		blocks_needed += 1;
	}
	size_logical = blocks_needed *THREADBLOCK_SIZE;

	double * host_buffer = new double[size];
	double * gpu_buffer_src = 0;
	double * gpu_buffer_dest = 0;

	for(int i=0;i<size;i++) {
		host_buffer[i] = 3.0;
	}

	cout << "Hi from scan test (one element at a time and REV)" << endl;
	cudasafe(hipMalloc((void**)&(gpu_buffer_src),size * sizeof(double)), 			"while mallocing on the gpu: __FILE__ __LINE__");
	cudasafe(hipMalloc((void**)&(gpu_buffer_dest),size * sizeof(double)), 			"while mallocing on the gpu: __FILE__ __LINE__");

	// copy data to gpu array
	hipMemcpy(gpu_buffer_src, host_buffer,  sizeof(double)*size, hipMemcpyHostToDevice); 

	for(int i=0;i<size;i++) {
		host_buffer[i] = 0.0;
	}

	// this is linear 1-d grid array.
	dim3 dimBlock(size);
	dim3 dimGrid(blocks_needed);
	
	// call the kernal...
	// it requires an array that is power of 2.
	cout << "Calling kddernal with " << size << " and " << blocks_needed << endl;
	//scanInclusiveShared_one_at_a_time_rev<<<dimGrid, dimBlock>>>( gpu_buffer_dest, gpu_buffer_src, size);
	scanInclusiveShared_one_at_a_time<<<dimGrid, dimBlock>>>( gpu_buffer_dest, gpu_buffer_src, size);
	cutilCheckMsg("Fuck up");

	// copy results back to host's fortran array.
	//hipMemcpy(parts.species[i].x, parts.species[i].cudaX,  sizeof(double)*parts.species[i].num_par_max, hipMemcpyDeviceToHost); 
	hipMemcpy(host_buffer, gpu_buffer_dest, sizeof(double)*size, hipMemcpyDeviceToHost); 
	for(int i=0;i<size;i++) {
		if (i%16==0) cout << endl;
		cout << host_buffer[i] << " ";
	}
	cout << endl;
}

void scan_test_4_at_a_time() {
	int size = 64;
	int size_logical = 64;

	// cheat for now and bump up mem size so eveything is even.
	int blocks_needed = size / (4*THREADBLOCK_SIZE);
	if((size % (4*THREADBLOCK_SIZE)) != 0) {
		blocks_needed += 1;
	}
	size_logical = blocks_needed * 4*THREADBLOCK_SIZE;

	double * host_buffer = new double[size];
	double * gpu_buffer_src = 0;
	double * gpu_buffer_dest = 0;

	for(int i=0;i<size;i++) {
		host_buffer[i] = 1.0;
	}

	cout << "Hi from scan test!" << endl;
	cudasafe(hipMalloc((void**)&(gpu_buffer_src),size * sizeof(double)), 			"while mallocing on the gpu: __FILE__ __LINE__");
	cudasafe(hipMalloc((void**)&(gpu_buffer_dest),size * sizeof(double)), 			"while mallocing on the gpu: __FILE__ __LINE__");

	// copy data to gpu array
	hipMemcpy(gpu_buffer_src, host_buffer,  sizeof(double)*size, hipMemcpyHostToDevice); 

	for(int i=0;i<size;i++) {
		host_buffer[i] = 0.0;
	}

	// this is linear 1-d grid array.
	dim3 dimBlock(size);
	dim3 dimGrid(blocks_needed);
	
	// call the kernal...
	cout << "Calling kernal with " << size << " and " << blocks_needed << endl;
	scanInclusiveShared<<<dimGrid, dimBlock>>>( (double4 *) gpu_buffer_dest, (double4 *) gpu_buffer_src, size);
	cutilCheckMsg("Fuck up");

	// copy results back to host's fortran array.
	//hipMemcpy(parts.species[i].x, parts.species[i].cudaX,  sizeof(double)*parts.species[i].num_par_max, hipMemcpyDeviceToHost); 
	hipMemcpy(host_buffer, gpu_buffer_dest, sizeof(double)*size, hipMemcpyDeviceToHost); 
	for(int i=0;i<size;i++) {
		if (i%20==0) cout << endl;
		cout << host_buffer[i] << " ";
	}
	cout << endl;
}
//__constant__ int NB;
//__constant__ double density_np;


/*
#######################################################################################################
#######################################################################################################
	6th attempt..... try to save some shared mem
#######################################################################################################
#######################################################################################################
*/

// This is the unsplit evaluation step.
 __device__ inline void eval_f_slopey_v6(double * fc, double * dest, double * logglam, const double *vr,
							//double const &_vr, double  const &_vrm1, double  const &_vrp1, 
							const uint size, const int padded_size, const int debug, volatile double *s_Data, const uint pos, const int eval_ind) {

	// these should be a 'synch' called before this routine is invoked.
	// the synch is needed bcuase data needs to be shared between the threads via the shared mem..
	double _vr = vr[id];
	double _vrm1 = 0;
	double _vrp1 = 0;
	if(id > 0) _vrm1 = vr[id-1];
	if(id < size-1) _vrp1 = vr[id+1];



	// load our data value in from fc (the shared mem chached input data)
	__syncthreads();
	
	double data = fc[id];
	double datam1 = 0.0;					// TODO: this needs to be fixed.I just wanna allocate the buffer to be a bit bigger do pos-1 won't seg fault at 0.
	if(id > 0) datam1 = fc[id-1];			

	//printf("%e %e %e %e %e\n",_vr, _vrm1,_vrp1,data, datam1)  ;
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;//*(_vr - _vrm1);
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;

	double final_I4 = scan1Inclusive(reduce_me, s_Data, padded_size);
	

																					//if(bi eval_num id == 0) printf("\n\tI4[n] inital: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ", id, final_I4);
	t = .5*_vr*_vr;
	tm1 = .5*_vrm1*_vrm1;
	reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	double final_I2 = scan1Inclusive(reduce_me, s_Data, padded_size);
																					//if(bi eval_num id == 0) printf("\n\tI2[n] inital: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ", id, final_I2);
	
	t = .5*_vrp1;
	tm1 = .5*_vr;
	reduce_me = t*fc[id+1] + tm1*data;
	

	reduce_me *= (_vrp1 - _vr);
	if(id == 31) reduce_me = 0.0;
																					//if(bi eval_num id == 0) printf("\n\tJ1 input: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ", id, reduce_me);


	double final_J1 = scan1Inclusive_rev(reduce_me, s_Data, padded_size);
	__syncthreads();
	
																					//if(	bi eval_num id == 0) printf("\n\tJ1[n] input: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ", id, final_J1);

	if( id  == (size-1)) {
		*logglam = LOGee_cuda(4.0*M_PI*final_I2, 4.0*M_PI*final_I4);
																					//if(	bi eval_num) printf("\n\tLoglam\n\t\t%e", (*logglam));
	}
					
	final_I2 *= final_J1;
	
	final_I2 *= -3.0;
	final_I4 += _vr*_vr*_vr * final_J1;
	final_I4 *= data;
	final_I4 += final_I2;


	
	__syncthreads();
																					//if(	bi eval_num id == 0) printf("\n\tG inital: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ",id,  final_I4);


	if(threadIdx.x < NB) {
		final_I4 = G_cuda(threadIdx.x, fc, vr, final_J1);
	}
																					//if(	bi eval_num id == 0) printf("\n\tG after bound: \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ", id, final_I4);

	// add another divergencew to take care of first cell...
	if(threadIdx.x == 0) {
		// store answer in u
		reduce_me = -1.0 * final_I4;
		reduce_me *= 2.0 / (vr[0]*vr[0]);
		dest[0] = reduce_me;
	}


	__syncthreads();
	s_Data[id] = final_I4;
	tm1 = s_Data[id+1];		// um1 is now I4[n+1]

	final_I4 -= tm1;			// I4[n] -= I4[n+1];

																					//if(	bi eval_num id == 0) printf("\n\tFirst 14 diff \n\t\t");
																					//if(bi eval_num id < 5) printf("%d:%e ",id,  final_I4);

	//u = Pn[id];
	
	t = ((_vrp1-_vr)*(_vrp1+_vr))/2.0; ///2.0/((_vrp1-_vr)*(_vrp1+_vr))
	
	//__syncthreads();
	if(t > 0)
		final_I4 /= t;				// I4[n] *= Pn[n];
																					//if(bi eval_num id == 0 && debug==1) printf("\n\tAfter Pn[n]:\n\t\t");
																					//if(bi eval_num id < 5 && debug==1) printf("%d:%e ", id, final_I4);
	s_Data[id] = final_I4;
	tm1 = s_Data[id+1];		// um1 is now I4[n+1]

	// this is terrible.. everyone will have to wait till this is done.
	if(threadIdx.x == 0) {
		reduce_me -= final_I4;
		reduce_me *= 2.0/(_vr*_vr*_vrp1);
		dest[0] = reduce_me;
	}
	__syncthreads();


	final_I4 -= tm1;			// I4[n] -= I4[n+1];
																					//if(bi eval_num id == 0 && debug==1) printf("\n\tSecond I4 diff\n\t\t");
																					//if(bi eval_num id < 5 && debug==1) printf("%d:%e ", id, final_I4);
	//u = Qn[id+1];

//1.0 / (vr[i+1]*vr[i+1]*(vr[i+2]-vr[i])/2.0);



	//t = ((_vrp1 - _vrm1)*_vr*_vr)/2.0;    //2.0/((_vrp1 - _vrm1)*_vr*_vr);
	t = (_vrp1*_vrp1*(vr[id+2]-_vr))/ 2.0;
	if(t > 0)
		final_I4 /= t;				// final_I4 = Qn[n+1]*I4[n];
	__syncthreads();

							 														//if(bi eval_num id == 0 && debug==1) printf("\n\tBefore Norm\n\t\t");
																					//if(bi eval_num id < 5 && debug==1) printf("%d:%e ", id, final_I4);		
	// TODO: prolly should put log ee in a register otr shared mem.
	// dp the normalization...

	final_I4 *= (*logglam);
	final_I4 *= c_kpre;

	dest[id+1] = final_I4;

	if(threadIdx.x == 0) {
		dest[0] *= (*logglam) *c_kpre;
	}
							 														//if(bi eval_num id == 0 && debug==1) printf("\n\After\n\t\t");
																					//if(bi eval_num id < 5 && debug==1) printf("%d:%e ", id, final_I4);		

}


__device__  double G_cuda_v6(const int n, 
							const double data0, const double data1, const double datan, 
							double vr0,double vr1, double vrn,
							double J1) {
	double i2s, i4s;
	// right now, everyone reads f00 and passes it in a register.. I think shared broadcasting may be faster.
	// read (vr0*vr0)/(vr1*vr1) from constant mem (which also boradcasts)
	
	double f00( (data0 - data1*(vr0*vr0)/(vr1*vr1))/ (1.0 - (vr0*vr0)/(vr1*vr1)) );
	i2s = f00*pow(vrn,3)/3.0 + (data1-f00)*pow(vrn,5)/(vr1*vr1)*0.2;
	i4s = f00*pow(vrn,5)*0.2 + (data1-f00)*pow(vrn,7)/(vr1*vr1*7.0);
	return datan*i4s + (pow(vrn,3)*datan-3.0*i2s) * J1;
	/*
	double i2s, i4s;
	double f00( (fin[0] - fin[1]*(vr[0]*vr[0])/(vr[1]*vr[1]))/ (1.0 - (vr[0]*vr[0])/(vr[1]*vr[1])) );
	//printf("\tf00 %e\n", f00);
	i2s = f00*pow(vr[n],3)/3.0 + (fin[1]-f00)*pow(vr[n],5)/(vr[1]*vr[1])*0.2;
	//printf("\ti2s %e\n", i2s);
	i4s = f00*pow(vr[n],5)*0.2 + (fin[1]-f00)*pow(vr[n],7)/(vr[1]*vr[1]*7.0);
	//printf("\ti4s %e\n", i4s);
	return fin[n]*i4s + (pow(vr[n],3)*fin[n]-3.0*i2s) * J1;*/

}



//#define DBROOT blockIdx.x == 0 && threadIdx.x==0 && debug == 1

template <unsigned int rk4_step>
 __global__ void rk4__ee_v6_sortee_PHYSCO(	double * __restrict__ fin, 
											double * __restrict__ trial_input_vector_to_use_in_evalation, 
											double * __restrict__ y_new, 
											const double * __restrict__ vr, 
											double * __restrict__ cell_data__loglam, 
											double * __restrict__ cell_data__precomp1,
											double * __restrict__ cell_data__precomp2,
											double * __restrict__ cell_data__precomp3,
											uint size, int padded_size, int debug, double h, int numh, double rk4_trial_factor, double rk4_factor) {	
	
	// declare needed shared mem.
	//__shared__ double swap_space[THREADBLOCK_SIZE];	
	__shared__ double swap_space[128];	
	__shared__ double zeroth;
	
																								LOG_ARRAY_INT("Reviving in physvo at RK4 step rk4_step with pos: ", debug, "\t");
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double final_I4 = cell_data__precomp1[pos];
	double _vr = vr[id];
	double _vrp1 = vr[id+1];	// we are ok with _vr because it havily zero padded wht only 1 copy in whole..
	double _vrp2 = vr[id+2];
	
	double final_I4_p1 = 0.0;
	if(id < (size-1)) {
		final_I4_p1 = cell_data__precomp1[pos+1];
	}
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
																								LOG_ARRAY("final I4 (results from prev. steps loaded from global mem):", final_I4, "\t");
																						
	
	// take care of the zero compoenent...
	if(threadIdx.x == 0) {
		zeroth = -1.0 * final_I4;
		zeroth *= 2.0 / (_vr*_vr);
	}
	
	final_I4 -= final_I4_p1;
																								LOG_ARRAY1("Result of first final_I4 diff:", zeroth, final_I4, "\t");
	
	
	double t = ((_vrp1-_vr)*(_vrp1+_vr))/2.0;  //(this is p[n])
	if(t > 0)
		final_I4 /= t;
																								LOG_ARRAY("After p[n]: ", final_I4, "\t");
	
	// This is Find DDG/(vDv)
	/*t= ((_vrp2-_vrp1)*(_vrp2+_vrp1))/2.0;  //(this is p[n+1])
	if(t > 0)
		final_I4 /= t;	
	__syncthreads();
	*/
	
	// Now take a forward derivitive (  I4[n] -= I4[n+1] )
	swap_space[id] = final_I4;
	__syncthreads();
	final_I4 -= swap_space[id+1];
																								LOG_ARRAY("After second I4 diff: ", final_I4, "\t");
	
	
	// final_I4 = Qn[n+1]*I4[n];
	t = (_vrp1*_vrp1*(_vrp2-_vr))/ 2.0;
	if(t > 0)
		final_I4 /= t;
	__syncthreads();
																								LOG_ARRAY("tBefore Noralization ", final_I4, "\t");
	
	final_I4 *= c_kpre;  // finishing the normalizing process..
																								LOG_ARRAY1("tFINAL EVAL ", rk4_trial_factor, final_I4, "\t");
	
	
	// So final_I4 contains the results from an evaulation of the F function of rk4.
	// But there is a slight wrinkle... it holds the result not of this 
	// position in the array, but the n+1 space in the array.
	// KILL THIS... it's just for debugging.
	t =  final_I4*rk4_trial_factor;
																								LOG_ARRAY1("eval*factor h/2: ", rk4_trial_factor, t, "\t");
	
	
	if(id < (size-1)) {
		/* use the answer we just computed and combine it with the original input vector
		   to use as the inital evulation input vector for next rk4 substep. */
		t = fin[pos+1];
		//t+= final_I4*rk4_trial_factor;
		//trial_input_vector_to_use_in_evalation[pos] = t
		trial_input_vector_to_use_in_evalation[pos+1] = t + final_I4*rk4_trial_factor;
		
		/* Also, we need to add this step into the overall answer all rk4 steps.
		   In the case of the 1st evaulation step, we need to prime the y_new vector.. so ass the result from this
		   step to the original input vector ams write to 'ynew'. 
		   Otherwise we simply add the result of this step to 'ynew'*/
		if(rk4_step == 1) {
			y_new[pos+1] = t + (final_I4*rk4_factor);
		} else if(rk4_step == 4) {
			t = y_new[pos+1];
			t += (final_I4*rk4_factor);
			fin[pos+1] = t;
		} else {
			t = y_new[pos+1];
			t += (final_I4*rk4_factor);
			y_new[pos+1] = t;
		}
	}
	__syncthreads();
	
	
	// I can do better then this..... maybe move it up to the last thread?
	// This is occuring because the answer in a thread is really the final answer for the [n+1] position..
	// so the zeroth thread gets left out... to really preserve symmetry, this calculation should really go into
	//   the last thread.. since that thread is not doing any usefull work (it's result get thrown out).
	if(threadIdx.x == 0) {
		zeroth -= swap_space[0]; // final_I4;
		zeroth *= 2.0/(_vr*_vr*_vrp1) * c_kpre;
		
																								LOG_ROOT("\tFinal 0th component this eval:        %e\n\t\t", zeroth);
		t = fin[pos];
		trial_input_vector_to_use_in_evalation[pos] = t + zeroth*rk4_trial_factor;
		if(rk4_step == 1) {
			y_new[pos] = fin[pos] + (zeroth*rk4_factor);
		} else if(rk4_step == 4) {
			t = y_new[pos];
			t += (zeroth*rk4_factor);
			fin[pos] = t;
		} else {
			t = y_new[pos];
																								LOG_ROOT("\tSaved 0th val before timestep update: %.8e \n\t\t", t);
			t += (zeroth*rk4_factor);
																								LOG_ROOT("\tSaved 0th val after timestep update:  %.8e \n\t\t", t);
			y_new[pos] = t;
		}
	}
	
	//__syncthreads();
	// KILL THIS... it's just for debugging.
	//t = trial_input_vector_to_use_in_evalation[pos];
	//																							LOG_ARRAY("The Starting values for next RK4 itteration sucker", t, "\t");
	//
	// If this is the 4th (and final) eval step, then copy results
	// to the results (input) buffer. [This is just copying.. so we can 
	// assume that every thread[n] has data for position[n]]
	// NOTE: this 'if(rk4_step==4)' is templated... so has no run-time cost.
	//if(rk4_step==4) {
	//	__syncthreads();
	//	t = y_new[pos];
	//	fin[pos] = t;
	//																							LOG_ROOT("-----------------------------------\n\t----------- Done with an RK4 Eval -----------\n-----------------------------------\n", t);
	//}
																								LOG_ARRAY("The current values of work-in-progress RK4 solution\n", t, "\t");
 }

 
/*
//    split version1
//
*/

// Shared mem for rk4 stuff..
// explicit:gl
//	 	1 global read (double)f
//	f 	1 global write (double)
//
#define SynchThreads __syncthreads();

__device__ inline void ass_fuck1( const int padded_size,  const uint pos, double * shared_buffer, const double vr, const double vrm1, const double ddata, const double ddatam1, double * __restrict__ out) {
	double t = .5*vr*vr;
	double tm1 = .5*vrm1*vrm1;
	double reduce_me = t*ddata + tm1*ddatam1;
	reduce_me *= (vr - vrm1); 
	if(id == 0) reduce_me = 0.0;
	
	out[pos] = scan1Inclusive(reduce_me, shared_buffer, padded_size);
}
__device__ inline void ass_fuck2( const int padded_size, const uint pos, double * shared_buffer, const double vr, const double vrm1, const double ddata, const double ddatam1, double * __restrict__ out) {
	double t = .5*vr*vr*vr*vr;
	double tm1 = .5*vrm1*vrm1*vrm1*vrm1;
	double reduce_me = t*ddata + tm1*ddatam1;
	reduce_me *= (vr - vrm1); 
	if(id == 0) reduce_me = 0.0;
	
	out[pos] = scan1Inclusive(reduce_me, shared_buffer, padded_size);
}

template <unsigned int test_type>
__global__ void test_forward_prefix_scans(double * buffer) {
	if (test_type == 0) {
		__shared__ double s_Data[132];
		double shit = 1.0;
		//printf("FUCK 0 \n");
		double crap = warpScanInclusiveSimpleWarp(shit, s_Data, 128) ;
		buffer[threadIdx.x] = crap;
	} else if (test_type == 1) {
		__shared__ double s_Data[132];
		double shit = 1.0;
		//printf("FUCK 1 \n"); 
		double crap = warpScanInclusiveSimple(shit, s_Data, 128) ;
		buffer[threadIdx.x] = crap;
	} else if (test_type == 2) {
		__shared__ double s_Data[132];
		double shit = 1.0;
		//printf("FUCK 2 \n");
		double crap = warpScanInclusiveSimpleWarp_rev(shit, s_Data, 128) ;
		buffer[threadIdx.x] = crap;
		//printf("%d %e \n", threadIdx.x, crap);
	} else {
		
		__shared__ double s_Data[2 * THREADBLOCK_SIZE];
		double shit = 1.0;
		//printf("FUCK 3 \n");
		double crap = scan1Inclusive(shit, s_Data, 128);
		buffer[threadIdx.x] = crap; 
	}
}


__global__ void rk4__ee_v6_sortee_FAGGOT_split3__1(	double * __restrict__ fin, double * __restrict__ dest, const double * __restrict__ vr, 
								    double * __restrict__ cell_data__loglam, 
									double * __restrict__ cell_data__precomp1,
									double * __restrict__ cell_data__precomp2,
									double * __restrict__ cell_data__precomp3,
									double * __restrict__ cell_data__precomp4,
									uint size, int padded_size, int debug, double h, int numh,
									const double * __restrict__ U4,
									const double * __restrict__ U4m1,
									const double * __restrict__ U2,
									const double * __restrict__ U2m1) {	
	
	//__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	__shared__ double s_Data[128];
	//__shared__ double logLambda;
	
	// Note: when comparing values in this funtion with the CPU e-e collsion code, keep in mind these values will be different
	//  by a factor of LogLambda (in the cpu code LogLombda is multiplied in the last step.. in the GPU code, it is mulitped eailier in the process)
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];										/* get momentum basis value for this cell. */
	double _vrm1 = 0;											/* get vr for point one behind. */
	if(id > 0) _vrm1 = vr[id-1];
	//double _vrp1 = 0;											/* get vr for point one ahead. */
	//if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];										/* get data point. */
	double datam1 = 0.0;										/* get data point for one position behind. */
	if(id > 0) datam1 = fin[pos-1];
	//double datap1 = 0.0;										/* get data point for one position ahead. */
	//if(id < size-1) datap1 = fin[pos+1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
																								LOG_ARRAY("Incoming Data to New Eval Step", data, "\t"); 
	
	/* now set up the calculation */
	
	/*
		// try functions...
		ass_fuck1(padded_size, pos, s_Data, _vr, _vrm1, data, datam1, cell_data__precomp4);
		//ass_fuck2(padded_size,pos, s_Data, _vr, _vrm1, data, datam1, cell_data__precomp1);
		double t = .5*_vr*_vr*_vr*_vr;
		double tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;
		double reduce_me = t*data + tm1*datam1;
		reduce_me *= (_vr - _vrm1); 
		if(id == 0) reduce_me = 0.0;
		cell_data__precomp1[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	*/
	
	/*
	// does the 1,2 clac at once using prcomed values (uses 19 registers)
	double t = U2[threadIdx.x];
	double tm1 = U2m1[threadIdx.x];
	double reduce_me = t*data + tm1*datam1;
	if(id == 0) reduce_me = 0.0;
	cell_data__precomp4[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	
	t = U4[threadIdx.x];
	tm1 = U4m1[threadIdx.x];
	reduce_me = t*data + tm1*datam1;
	if(id == 0) reduce_me = 0.0;
	cell_data__precomp1[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	*/
	
	
	/*
	// does the 1,2 calc at once. tries to resue variables (uses 28 fucking registers)
	double t = .5*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	
	cell_data__precomp4[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	
	
	t *= _vr*_vr;
	tm1 *= _vrm1*_vrm1;
	reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	cell_data__precomp1[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	*/
	
	/*
	// does the 1,2 calc at once. tries to resue variables another way(uses 28 fucking registers)
	double t = .5*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	//cell_data__precomp4[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	cell_data__precomp4[pos] = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	*/
	
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 =.5*_vrm1*_vrm1*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	//cell_data__precomp1[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	cell_data__precomp1[pos] = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	
	
	//t = scan1Inclusive(reduce_me, s_Data, padded_size);
	//cell_data__precomp4[pos] = 43.9;
	
	
	//cell_data__precomp1[pos] = 42.0;
	
	
	/*
	//  does the 1,2 calc at once. renames variables. (uses 28 fucking registers)
	
	double t = .5*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	cell_data__precomp4[pos] = scan1Inclusive(reduce_me, s_Data, padded_size);
	
	double fuck_nvidas_shitty_compilier_t = .5*_vr*_vr*_vr*_vr;
	double fuck_nvidas_shitty_compilier_tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;
	double fuck_nvidas_shitty_compilier_reduce_me = fuck_nvidas_shitty_compilier_t*data + fuck_nvidas_shitty_compilier_tm1*datam1;
	fuck_nvidas_shitty_compilier_reduce_me *= (_vr - _vrm1);
	if(id == 0) fuck_nvidas_shitty_compilier_reduce_me = 0.0;
	cell_data__precomp1[pos] = scan1Inclusive(fuck_nvidas_shitty_compilier_reduce_me, s_Data, padded_size);
	*/
	
	/*
	// just does 1 calc then leaves..
	// now set up the calculation 
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	
	// do the 'integral' (sum) 
	double final_I4 = scan1Inclusive(reduce_me, s_Data, padded_size);
																								LOG_ARRAY("I4[n] inital", final_I4, "\t"); 
	
	
	//save result in global (persistent) mem for next subkernal 
	cell_data__precomp1[pos] = final_I4;
																								LOG_ROOT("------------END OF SPLIT3_1--------------", debug);
	*/
	
	/*
	//newer, low shared mem version of 'just one calc then leave'
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 =.5*_vrm1*_vrm1*_vrm1*_vrm1;
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	
	//double final_I4 = scan1Inclusive(reduce_me, s_Data, padded_size);
	double final_I4 = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
																								LOG_ARRAY("I4[n] inital", final_I4, "\t"); 
	
	
	//save result in global (persistent) mem for next subkernal 
	cell_data__precomp1[pos] = final_I4;
																								LOG_ROOT("------------END OF SPLIT3_1--------------", debug);
	*/
	
}

__global__ void rk4__ee_v6_sortee_FAGGOT_split3__2(	double * __restrict__ fin, double * __restrict__ dest, const double * __restrict__ vr, 
								    double * __restrict__ cell_data__loglam, 
									double * __restrict__ cell_data__precomp1,
									double * __restrict__ cell_data__precomp2,
									double * __restrict__ cell_data__precomp3,
									double * __restrict__ cell_data__precomp4,
									uint size, int padded_size, int debug, double h, int numh) {	
	
	//__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	__shared__ double s_Data[128];
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];										/* get momentum basis value for this cell. */
	double _vrm1 = 0;											/* get vr for point one behind. */
	if(id > 0) _vrm1 = vr[id-1];
	//double _vrp1 = 0;											/* get vr for point one ahead. */
	//if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];										/* get data point. */
	double datam1 = 0.0;										/* get data point for one position behind. */
	if(id > 0) datam1 = fin[pos-1];
	//double datap1 = 0.0;										/* get data point for one position ahead. */
	//if(id < size-1) datap1 = fin[pos+1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
	
	
	/* now set up the calculation */
	double t = .5*_vr*_vr;
	double  tm1 = .5*_vrm1*_vrm1;
	double  reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	
	/* do the 'integral' (sum) */
	double final_I2 = 	warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	//double final_I2 = scan1Inclusive(reduce_me, s_Data, padded_size);
	
	/* save result in global (persistent) mem for next subkernal */
	cell_data__precomp4[pos] = final_I2;
																								LOG_ARRAY("I2[n] inital", final_I2, "\t"); 
																								LOG_ROOT("------------END OF SPLIT3_2--------------", debug);
	
}

__global__ void rk4__ee_v6_sortee_FAGGOT_split3__2_5(	double * __restrict__ fin, double * __restrict__ dest, const double * __restrict__ vr, 
								    double * __restrict__ cell_data__loglam, 
									double * __restrict__ cell_data__precomp1,
									double * __restrict__ cell_data__precomp2,
									double * __restrict__ cell_data__precomp3,
									double * __restrict__ cell_data__precomp4,
									double * __restrict__ cell_data__precomp5,
									uint size, int padded_size, int debug, double h, int numh) {
	
	//__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	__shared__ double s_Data[128];
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];									/* get momentum basis value for this cell. */
	double _vrp1 = 0;										/* get vr for point one ahead. */
	if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];									/* get data point. */
	double datap1 = 0.0;									/* get data point for one position ahead. */
	if(id < size-1) datap1 = fin[pos+1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
	
	/* now set up the calculation */
	double t = .5*_vrp1;
	double tm1 = .5*_vr;
	double reduce_me = t*datap1 + tm1*data;
	reduce_me *= (_vrp1 - _vr);
	if(id >= (size-1)) reduce_me = 0.0;
	
																								LOG_ARRAY("J1 input", reduce_me, "\t"); 
	/* do the 'integral' (sum) note: this is reversed. */
	double final_J1 = warpScanInclusiveSimpleWarp_rev(reduce_me, s_Data, padded_size);
	//double final_J1 = scan1Inclusive_rev(reduce_me, s_Data, padded_size);
	
	/* save result in global (persistent) mem for next subkernal */
	cell_data__precomp5[pos] = final_J1;
	
}

__global__ void rk4__ee_v6_sortee_FAGGOT_split3__3(	double * __restrict__ fin, double * __restrict__ dest, const double * __restrict__ vr, 
								    double * __restrict__ cell_data__loglam, 
									double * __restrict__ cell_data__precomp1,
									double * __restrict__ cell_data__precomp2,
									double * __restrict__ cell_data__precomp3,
									double * __restrict__ cell_data__precomp4,
									double * __restrict__ cell_data__precomp5,
									uint size, int padded_size, int debug, double h, int numh) {
	
	__shared__ double logLambda;
	//__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];									/* get momentum basis value for this cell. */
	//double _vrm1 = 0;										/* get vr for point one behind. */
	//if(id > 0) _vrm1 = vr[id-1];
	//double _vrp1 = 0;										/* get vr for point one ahead. */
	//if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];									/* get data point. */
	//double datap1 = 0.0;									/* get data point for one position ahead. */
	//if(id < size-1) datap1 = fin[pos+1];
	//double datam1 = 0.0;									/* get data point for one position behind. */
	//if(id > 0) datam1 = fin[pos-1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
	
	/*
	double t = .5*_vrp1;
	double tm1 = .5*_vr;
	double reduce_me = t*datap1 + tm1*data;
	
	
	reduce_me *= (_vrp1 - _vr);
	if(id >= (size-1)) reduce_me = 0.0;
																								LOG_ARRAY("J1 input", reduce_me, "\t"); 
	
	double final_J1 = scan1Inclusive_rev(reduce_me, s_Data, padded_size);
	__syncthreads();
																								LOG_ARRAY("J1[n] processed", final_J1, "\t");
	*/
	
	/* recall the values calculated by the previous sub kernals */
	double final_I4 = cell_data__precomp1[pos];
	double final_I2 = cell_data__precomp4[pos];
	double final_J1 = cell_data__precomp5[pos];
	
																								LOG_ARRAY("Data from Split-Step-1 coming in Split-Step-3: ", final_I4, "\t");
																								LOG_ARRAY("Data from Split-Step-2 coming in Split-Step-3: ", final_I2, "\t");
	
	if(id == size-1) {
        //double logLambda;
		logLambda = 2;
        double ne = 4.0*M_PI*final_I2; double Te = 4.0*M_PI*final_I4;
		// if the density is positive
        if (ne > 0.000000001) {
            Te /= (3.0*ne);
            Te *= 511000; // Temperature in eV
            ne *= density_np;

            Te = log(Te); 
            ne = log(ne);
            logLambda = 23.5 - 0.5*ne + 1.25*Te - sqrt(0.00001+0.0625*(Te-2.0)*(Te-2.0));

            if (logLambda < 2.0) logLambda = 2;
        }
		//logLambda = LOGee_cuda(4.0*M_PI*final_I2,4.0*M_PI*final_I4);
	}
																								LOG_ROOT("\tLog Lambda\n\t\t%e", 	logLambda);
	__syncthreads();
	
	
	final_I4 = data*final_I4 + _vr*_vr*_vr * final_J1 * data - 3.0*final_J1*final_I2;
	
																								LOG_ARRAY("G intital", final_I4, "\t");
	/* fixup the lower cells (filter them) to fix up issues iwth shpherical harmonincs */
	if (id < NB) { 
		final_I4 = G_cuda_v6(	id, fin[blockIdx.x*blockDim.x], 
								fin[blockIdx.x*blockDim.x + 1], 
								data, vr[0], vr[1], _vr, final_J1);
	}
																								LOG_ARRAY("G after lower boundry conditions applied", final_I4, "\t");
	
	final_I4 *= logLambda;
	
	/* save result in global (persistent) mem for next subkernal */
	cell_data__precomp1[pos] = final_I4;
																								LOG_ROOT("------------END OF SPLIT3_3--------------", debug);
	
}

__global__ void rk4__ee_v6_sortee_FAGGOT(	double * __restrict__ fin, 
											double * __restrict__ trial_input_vector_to_use_in_evalation, 
											const double * __restrict__ vr, 
											double * __restrict__ cell_data__loglam, 
											double * __restrict__ cell_data__precomp1,
											double * __restrict__ cell_data__precomp2,
											double * __restrict__ cell_data__precomp3,
											uint size, int padded_size, 
											int debug, double h, int numh) {	
	
	//__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	__shared__ double s_Data[128];
	__shared__ double logLambda;
	// Note: when comparing values in this funtion with the CPU e-e collsion code, keep in mind these values will be different
	//  by a factor of LogLambda (in the cpu code LogLombda is multiplied in the last step.. in the GPU code, it is mulitped eailier in the process)
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];
	double _vrm1 = 0;
	double _vrp1 = 0;
	if(id > 0) _vrm1 = vr[id-1];
	if(id < size-1) _vrp1 = vr[id+1];
	
	__syncthreads();
	//if(rk4_eval_step==1) {
		double data = fin[pos];
		double datam1 = 0.0;					// TODO: this needs to be fixed.I just wanna allocate the buffer to be a bit bigger do pos-1 won't seg fault at 0.
		if(id > 0) datam1 = fin[pos-1];			
		double datap1 = 0.0;
		if(id < size-1) datap1 = fin[pos+1];
	//} else {
	//	data = trial_input_vector_to_use_in_evalation[id];
	//	datam1 = 0.0;					// TODO: this needs to be fixed.I just wanna allocate the buffer to be a bit bigger do pos-1 won't seg fault at 0.
	//	if(id > 0) datam1 = trial_input_vector_to_use_in_evalation[id-1];			
	//	datap1 = 0.0;
	//	if(id < size-1) datap1 = trial_input_vector_to_use_in_evalation[id+1];		
	//}
																					LOG_ARRAY("Incoming Data to New Eval Step", data, "\t"); 
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;//*(_vr - _vrm1);
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	
	//double final_I4 = scan1Inclusive(reduce_me, s_Data, padded_size);
	//double final_I4 = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	double final_I4 = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	
	
																					LOG_ARRAY("I4[n] inital", final_I4, "\t"); 
	t = .5*_vr*_vr;
	tm1 = .5*_vrm1*_vrm1;
	reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	__syncthreads(); // suspect.
	
	//double final_I2 = scan1Inclusive(reduce_me, s_Data, padded_size);
	double final_I2 = warpScanInclusiveSimpleWarp(reduce_me, s_Data, padded_size);
	
																					LOG_ARRAY("I2[n] inital", final_I2, "\t"); 

	if(id == size-1) {
		logLambda = LOGee_cuda(4.0*M_PI*final_I2,4.0*M_PI*final_I4);//*c_kpre;	
		//printf("\tLog lambda (args=%e,%e): %e", 4.0*M_PI*final_I2, 4.0*M_PI*final_I4, logLambda);
		//logLambda *= c_kpre;
	}

	//if(DBROOT) {
	//	printf("\npadding %e %e %e %e \n", fin[blockIdx.x* blockDim.x + 28],fin[blockIdx.x* blockDim.x + 29],fin[blockIdx.x* blockDim.x + 30],fin[blockIdx.x* blockDim.x + 31]);
	//}
	
	t = .5*_vrp1;
	tm1 = .5*_vr;
	reduce_me = t*datap1 + tm1*data;
	
	
	reduce_me *= (_vrp1 - _vr);
	//if(id == (size-1)) reduce_me = 0.0;
	// to be very safe for now.. relax later...
	if(id >= (size-1)) reduce_me = 0.0;
																					LOG_ARRAY("J1 input", reduce_me, "\t"); 


	//double final_J11 = scan1Inclusive_rev(reduce_me, s_Data, padded_size);
	double final_J1 = warpScanInclusiveSimpleWarp_rev(reduce_me, s_Data, padded_size);
																					LOG_ARRAY("J1[n] processed", final_J1, "\t");
/* 
	//
	// way 1 stop computation here ( Used 31 registers, 4096+0 bytes smem) these are nice numbers.
	//

	cell_data__precomp1[blockIdx.x] = final_I4;
	cell_data__precomp2[blockIdx.x] = final_I2;
	cell_data__precomp3[blockIdx.x] = final_J1;
*/
	//if( id  == (size-1)) {
	//	double logglam = LOGee_cuda(4.0*M_PI*final_I2, 4.0*M_PI*final_I4);				if(DBROOT) printf("\n\tLoglam\n\t\t%e", (*logglam));
	//}

	//
	// Also try doing a bit more computation over here..
	//
	final_I4 = data*final_I4 + _vr*_vr*_vr * final_J1 * data - 3.0*final_J1*final_I2;		LOG_ARRAY("G intital", final_I4, "\t");
																							
	
	if (id < NB) {
		final_I4 = G_cuda_v6(id, fin[blockIdx.x*blockDim.x], fin[blockIdx.x*blockDim.x + 1], data, vr[0], vr[1], _vr, final_J1);
	}
																							LOG_ROOT("\tLog Lambda\n\t\t%e", 	logLambda);																			

	///     Evaluate G assuming a parabolic f(v << vt)
	//for (int n(0); n < NB; ++n) { 
	//	I4[n] = G(n,fin);
	//}
																							LOG_ARRAY("G after lower boundry conditions applied", final_I4, "\t");
																							
	// this may not work... but pre normalize...
	
    final_I4 *= logLambda;


	//cell_data__precomp1[blockIdx.x] = final_I4;
	cell_data__precomp1[pos] = final_I4;
	__syncthreads();
}

/*
//    split eval into 2 functions!
//
*/
__global__ void rk4__ee_v6_sortee_FAGGOT_split2__1__v2(	double * __restrict__ fin,
														double * __restrict__ dest,
														const double * __restrict__ vr, 
														double * __restrict__ cell_data__loglam, 
														double * __restrict__ cell_data__precomp1,
														double * __restrict__ cell_data__precomp2,
														double * __restrict__ cell_data__precomp3,
														uint size, int padded_size, int debug, double h, int numh) {
	
	__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];										/* get momentum basis value for this cell. */
	double _vrm1 = 0;											/* get vr for point one behind. */
	if(id > 0) _vrm1 = vr[id-1];
	//double _vrp1 = 0;											/* get vr for point one ahead. */
	//if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];										/* get data point. */
	double datam1 = 0.0;										/* get data point for one position behind. */
	if(id > 0) datam1 = fin[pos-1];
	//double datap1 = 0.0;										/* get data point for one position ahead. */
	//if(id < size-1) datap1 = fin[pos+1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
	
																								LOG_ARRAY("Incoming Data to New Eval Step", data, "\t"); 
	double t = .5*_vr*_vr*_vr*_vr;
	double tm1 = .5*_vrm1*_vrm1*_vrm1*_vrm1;//*(_vr - _vrm1);
	double reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1);
	if(id == 0) reduce_me = 0.0;
	
	double final_I4 = scan1Inclusive(reduce_me, s_Data, padded_size);
																								LOG_ARRAY("I4[n] inital", final_I4, "\t"); 
	
	/* save result in global (persistent) mem for next subkernal */
	cell_data__precomp1[pos] = final_I4;
																								LOG_ROOT("------------END OF SPLIT1--------------", debug);
	
}

__global__ void rk4__ee_v6_sortee_FAGGOT_split2__2__v2(	double * __restrict__ fin, 
														double * __restrict__ dest,
														const double * __restrict__ vr, 
														double * __restrict__ cell_data__loglam, 
														double * __restrict__ cell_data__precomp1,
														double * __restrict__ cell_data__precomp2,
														double * __restrict__ cell_data__precomp3,
														uint size, int padded_size, int debug, double h, int numh) {	
	// --- restart and resume place boilerplate---------
	__shared__ double s_Data[2 * THREADBLOCK_SIZE];
	__shared__ double logLambda;
	
	/* ------------------------------------------------- */
	/* --- restart and resume place boilerplate--------- */
	uint pos = blockIdx.x* blockDim.x + threadIdx.x;
	
	double _vr = vr[id];										/* get momentum basis value for this cell. */
	double _vrm1 = 0;											/* get vr for point one behind. */
	if(id > 0) _vrm1 = vr[id-1];
	double _vrp1 = 0;											/* get vr for point one ahead. */
	if(id < size-1) _vrp1 = vr[id+1];
	
	double data = fin[pos];										/* get data point. */
	double datam1 = 0.0;										/* get data point for one position behind. */
	if(id > 0) datam1 = fin[pos-1];
	double datap1 = 0.0;										/* get data point for one position ahead. */
	if(id < size-1) datap1 = fin[pos+1];
	
	/* --- END restart and resume place boilerplate----- */
	/* ------------------------------------------------- */
	
	/* load previously saved result */
	double final_I4 = cell_data__precomp1[pos];
																								LOG_ARRAY("Data from Split-Step-1 coming in Split-Step-2: ", final_I4, "\t");
	
	
	/* now set up the calculation */
	double t = .5*_vr*_vr;
	double  tm1 = .5*_vrm1*_vrm1;
	double  reduce_me = t*data + tm1*datam1;
	reduce_me *= (_vr - _vrm1); 
	if(id == 0) reduce_me = 0.0;
	
	/* do the 'integral' (sum) */
	double final_I2 = scan1Inclusive(reduce_me, s_Data, padded_size);
																								LOG_ARRAY("I2[n] inital", final_I2, "\t"); 
	
	if(id == size-1) {
		logLambda = LOGee_cuda(4.0*M_PI*final_I2,4.0*M_PI*final_I4);
	}
	
	
	/* now set up the calculation */
	t = .5*_vrp1;
	tm1 = .5*_vr;
	reduce_me = t*datap1 + tm1*data;
	reduce_me *= (_vrp1 - _vr);
	// to be very safe for now.. relax later...
	if(id >= (size-1)) reduce_me = 0.0;
																								LOG_ARRAY("J1 input", reduce_me, "\t"); 
	/* do the 'integral' (sum) note: this is reversed. */
	double final_J1 = scan1Inclusive_rev(reduce_me, s_Data, padded_size);
	__syncthreads();
																								LOG_ARRAY("J1[n] calculated", final_J1, "\t");
	
	
	/* calculate the result we after by combining all the previous integral values! */
	final_I4 = data*final_I4 + _vr*_vr*_vr * final_J1 * data - 3.0*final_J1*final_I2;
																								LOG_ARRAY("G intital", final_I4, "\t");
	
	/* fixup the lower cells (filter them) to fix up issues iwth shpherical harmonincs */
	if (id < NB) {
		final_I4 = G_cuda_v6(	id, fin[blockIdx.x*blockDim.x],
								fin[blockIdx.x*blockDim.x + 1], 
								data, vr[0], vr[1], _vr, final_J1);
	}
	
																								LOG_ROOT("\tLog Lambda\n\t\t%e", 	logLambda);																			
																								LOG_ARRAY("G after lower boundry conditions applied", final_I4, "\t");
	final_I4 *= logLambda;
	
	/* save result in global (persistent) mem for next subkernal */
	cell_data__precomp1[pos] = final_I4;
																								LOG_ROOT("----------------END OF SPLIT2---------------", debug);
}

void test_scans(double * trash_buffer) {
	
	dim3 dimGrid_test(1);
	dim3 dimBlock_test(128);
	
	/*
	{ ScopeTimer ___t("scan_test_0");
		for(int i = 0; i < 1000;i++) {
			test_forward_prefix_scans<0><<<dimGrid_test, dimBlock_test>>>(trash_buffer);
		}
	}
	
	{ ScopeTimer ___t("scan_test_1");
		for(int i = 0; i < 1000;i++) {
			test_forward_prefix_scans<1><<<dimGrid_test, dimBlock_test>>>(trash_buffer);
		}
	}
	
	{ ScopeTimer ___t("scan_test_2");
		for(int i = 0; i < 1000;i++) {
			test_forward_prefix_scans<3><<<dimGrid_test, dimBlock_test>>>(trash_buffer);
		}
	}
	
	
	double timer0 = TimerOracle::summon()->get_result("scan_test_0")->dt;
	double timer1 = TimerOracle::summon()->get_result("scan_test_1")->dt;
	double timer2 = TimerOracle::summon()->get_result("scan_test_2")->dt;
	
	cout << "Method 0: " << timer0 << " ms" << endl;
	cout << "Method 1: " << timer1 << " ms" << endl;
	cout << "Method 2: " << timer2 << " ms" << endl;
	*/
	
	test_forward_prefix_scans<2><<<dimGrid_test, dimBlock_test>>>(trash_buffer);
}

// Good entry point... works..
void eval_rk4_v6 (	double * fc, double * dest, double *vr, 
							//double * U1, double *U1m1, double * U2, double *U2m1,
							//double * U3,
							//double * U4, double *U4m1,
							//double * Pn, double *Qn,
							double * __restrict__ cell_data__precomp1,   // these should be the size of the number of cells.*pr
							double * __restrict__ cell_data__precomp2,
							double * __restrict__ cell_data__precomp3,
							double * __restrict__ cell_data__precomp4,
							double * __restrict__ cell_data__precomp5,
							uint size, int padded_size, int debug, double h, int numh, int num_cells_x, int num_cells_y,
																const double * __restrict__ U4,
									const double * __restrict__ U4m1,
									const double * __restrict__ U2,
									const double * __restrict__ U2m1) {
	
	// I gotta clean up threse names... the ultimate answer is passed back to Host via the first argument (fc)
	//									this buffer islso the buffer that has the incoming data.
	//
	//									for this reason, it is not altered until the last part of an RK4 step.
	//
	
	// cheat for now and bump up mem size so eveything is even.
	int blocks_needed = size / (1*THREADBLOCK_SIZE);
	if((size % (1*THREADBLOCK_SIZE)) != 0) {
		blocks_needed += 1;
	}
	//
	
	// this is linear 1-d grid array.
	dim3 dimBlock(padded_size);
	dim3 dimGrid(num_cells_x*num_cells_y);
	
	double * yn = cell_data__precomp3;
	double * y_new = cell_data__precomp2;
	//numh = 1;  // for debugging.. so this loop runs only once.
	
	//cout << "Hi from eval test!" << padded_size << endl;
	
	for(int i =0;i<numh;i++) {
		if(false) { 
			//hipStream_t stream[3]; 
			//for (int i = 0; i < 3; ++i) 
			//	hipStreamCreate(&stream[i]);
			//cout << "Trying super streams! " << endl << blocks_needed << ":"<< padded_size << endl;
			if(true) {
				// 
				// first evaulation of the function (1/4th of an RK4 step) should read from (have 1st argument) fc (current function state)
				// after 1st evaluation, the next state to use is copied into the 'yn' variable.. so all other evals should be reading from this.
				// ynew hold the answer as it is axxumated fron eaxhg step (it's a sum)
				//
				// this stramge stucture exists because RK4 does 4  funtion evaulations.. at 'locations' that are relative to
				// the input... so the input (fc) needsd to exist unchanged (can't be overwritten) until very last part of an RK4 step.
				// so the anser must be accumated in another buffer and the the trial evaluation point in a third.
				//
		// RK4 eval 1
				rk4__ee_v6_sortee_FAGGOT_split3__1<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 1, h, numh, U4, U4m1, U2, U2m1);
				
				rk4__ee_v6_sortee_FAGGOT_split3__2<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 1, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__2_5<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 1, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__3<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 1, h, numh);
				//hipDeviceSynchronize();
				rk4__ee_v6_sortee_PHYSCO<1><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 1, h, numh, .5*h, h/6.0);
				
				// notice that the frist 2 args of 'faggor' are reverese relative to the previous invokations.
		// RK4 eval 2
				rk4__ee_v6_sortee_FAGGOT_split3__1<<< dimGrid, dimBlock>>>( yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 1, h, numh, U4, U4m1, U2, U2m1);
				
				rk4__ee_v6_sortee_FAGGOT_split3__2<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 2, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__2_5<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 2, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__3<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 2, h, numh);
				//hipDeviceSynchronize();
				rk4__ee_v6_sortee_PHYSCO<2><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 2, h, numh, .5*h, h/3.0);
				
				
		// RK4 eval 3
				rk4__ee_v6_sortee_FAGGOT_split3__1<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 1, h, numh, U4, U4m1, U2, U2m1);
				
				rk4__ee_v6_sortee_FAGGOT_split3__2<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 3, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__2_5<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 3, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__3<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 3, h, numh);
				//hipDeviceSynchronize();
				rk4__ee_v6_sortee_PHYSCO<3><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 3, h, numh, h, h/3.0);
				
				
		// RK4 eval 4
				rk4__ee_v6_sortee_FAGGOT_split3__1<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 1, h, numh, U4, U4m1, U2, U2m1);
				
				rk4__ee_v6_sortee_FAGGOT_split3__2<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														size, padded_size, 4, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__2_5<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 4, h, numh);
				rk4__ee_v6_sortee_FAGGOT_split3__3<<< dimGrid, dimBlock>>>(	yn, fc, vr, //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														cell_data__precomp4,
														cell_data__precomp5,
														size, padded_size, 4, h, numh);
				//hipDeviceSynchronize();
				rk4__ee_v6_sortee_PHYSCO<4><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 4, h, numh, h, h/6.0);
				
				
			} else {
				//cout << "Using split 2 version" << endl;
				
				// aternate version with 2 split threads...
		// RK4 eval 1
				rk4__ee_v6_sortee_FAGGOT_split2__1__v2<<< dimGrid, dimBlock>>>(	fc, yn, vr,  //fc, dest, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 1, h, numh);
				
				rk4__ee_v6_sortee_FAGGOT_split2__2__v2<<< dimGrid, dimBlock>>>(	fc, yn, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 1, h, numh);
				
				//hipDeviceSynchronize();
				
				rk4__ee_v6_sortee_PHYSCO<1><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 1, h, numh, .5*h, h/6.0);
				
		// RK4 eval 2
				rk4__ee_v6_sortee_FAGGOT_split2__1__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr,
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 2, h, numh);
				
				rk4__ee_v6_sortee_FAGGOT_split2__2__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 2, h, numh);
				
				//hipDeviceSynchronize();
				
				rk4__ee_v6_sortee_PHYSCO<2><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 2, h, numh, .5*h, h/3.0);
		// RK4 eval 3
				rk4__ee_v6_sortee_FAGGOT_split2__1__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr,
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 3, h, numh);
				
				rk4__ee_v6_sortee_FAGGOT_split2__2__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr, 
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 3, h, numh);
				
				//hipDeviceSynchronize();
				
				rk4__ee_v6_sortee_PHYSCO<3><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 3, h, numh, h, h/3.0);
		// RK4 eval 4
				rk4__ee_v6_sortee_FAGGOT_split2__1__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr,
														cell_data__precomp1,
														cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 4, h, numh);
				
				rk4__ee_v6_sortee_FAGGOT_split2__2__v2<<< dimGrid, dimBlock>>>(	yn, fc, vr,
														cell_data__precomp1,
				 										cell_data__precomp1,
														cell_data__precomp2,
														cell_data__precomp3,
														size, padded_size, 4, h, numh);
				
				//hipDeviceSynchronize();
				
				rk4__ee_v6_sortee_PHYSCO<4><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, cell_data__precomp1, 
												cell_data__precomp1,
												cell_data__precomp2,
												cell_data__precomp3,
												size, padded_size, 4, h, numh, h, h/6.0); 
				
			}
			//hipStreamDestroy(stream[0]);
			//hipStreamDestroy(stream[1]);
			//hipStreamDestroy(stream[2]);
		} else {
		// hipDeviceSynchronize();	
			
																												{ //ScopeTimer ___t("CUDA e*e collsion kernal launch time per RK4 advance");
		// RK4 eval 1
			rk4__ee_v6_sortee_FAGGOT	<<< dimGrid, dimBlock >>>(	fc, yn, vr, 
																	cell_data__precomp1,
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 1, h, numh); 
			
			rk4__ee_v6_sortee_PHYSCO<1> <<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, 
																	cell_data__precomp1, 
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 1, h, numh, 0.5*h, h/6.0); 
			
		// RK4 eval 2
			rk4__ee_v6_sortee_FAGGOT	<<< dimGrid, dimBlock >>>(	yn, fc, vr, 		// Note the reversed order of the 1st 2 arguments fc, yn
																	cell_data__precomp1,
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 2, h, numh); 
			
			rk4__ee_v6_sortee_PHYSCO<2> <<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, 
																	cell_data__precomp1, 
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 2, h, numh, .5*h, h/3.0); 
																					// 2
			rk4__ee_v6_sortee_FAGGOT	<<< dimGrid, dimBlock >>>(	yn, fc, vr, 		// Note the reversed order of the 1st 2 arguments fc, yn
																	cell_data__precomp1,
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 3, h, numh); 
			
		// RK4 eval 3
			rk4__ee_v6_sortee_PHYSCO<3>	<<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, 
																	cell_data__precomp1, 
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 3, h, numh, h, h/3.0);
																					// 3
			rk4__ee_v6_sortee_FAGGOT<<< dimGrid, dimBlock >>>(		yn, fc, vr, 		// Note the reversed order of the 1st 2 arguments fc, yn
																	cell_data__precomp1,
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
																	size, padded_size, 4, h, numh); 
			
		// RK4 eval 4
			rk4__ee_v6_sortee_PHYSCO<4><<< dimGrid, dimBlock >>>(	fc, yn, y_new, vr, 
																	cell_data__precomp1, 
																	cell_data__precomp1,
																	cell_data__precomp2,
																	cell_data__precomp3,
  																	size, padded_size, 4, h, numh, h, h/6.0); 
																					// 4
																													}
		}
	}
	
	cutilCheckMsg("An error occured while launching the kernal. Exiting.");
	//cout << "\tLaunched " << (numh*8) << " kernals for a " << numh << " consecutive RK4 step process.\n";
}
